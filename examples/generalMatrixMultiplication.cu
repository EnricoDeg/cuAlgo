#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <cuAlgo.hpp>

int main() {

	int N     = 1024;
	int M     = N;
	int K     = N;
	int T     = 32;
	int alpha = 1;
	int beta  = 0;

	int * A        = (int *)malloc(N * N * sizeof(int));
	int * B        = (int *)malloc(N * N * sizeof(int));
	int * C        = (int *)malloc(N * N * sizeof(int));
	int * solution = (int *)malloc(N * N * sizeof(int));

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j)
			A[j + i * N] = j;

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j)
			B[j + i * N] = i;

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j)
			C[j + i * N] = 1;

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j)
			solution[j + i * N] = 0;

	for (int m = 0; m < M; m += T) {
		for (int n = 0; n < N; n += T) {
			for (int k = 0; k < K; k += T) {

				const int minMt = std::min(m + T, M);
				const int minNt = std::min(n + T, N);
				const int minKt = std::min(k + T, K);

				for (int mt = m; mt < minMt; mt++) {
					for (int nt = n; nt < minNt; nt++) {
						for (int kt = k; kt < minKt; kt++) {
							solution[mt * M + nt] += A[mt * M + kt] * B[kt * K + nt];
						}
					}
				}
			}
		}
	}

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j)
			solution[j + i * N] = alpha * solution[j + i * N] + beta * C[j + i * N];

	int *d_A;
	check_cuda( hipMalloc(&d_A, N*N*sizeof(int)) );

	int *d_B;
	check_cuda( hipMalloc(&d_B, N*N*sizeof(int)) );

	int *d_C;
	check( hipMalloc(&d_C, N*N*sizeof(int)) );

	check_cuda( hipMemcpy ( d_A, A, (size_t)N*N*sizeof(int), hipMemcpyHostToDevice ) );

	check_cuda( hipMemcpy ( d_B, B, (size_t)N*N*sizeof(int), hipMemcpyHostToDevice ) );

	check_cuda( hipMemcpy ( d_C, C, (size_t)N*N*sizeof(int), hipMemcpyHostToDevice ) );

	for (int i = 0; i < 5; ++i)
		gMatMul(N, N, N, alpha, d_A, d_B, beta, d_C);

	check_cuda( hipMemcpy ( C, d_C, N*N*sizeof(int), hipMemcpyDeviceToHost ) );

	for(int i = 0; i < N; ++i)
		for (int j = 0; j < N ; ++j) {
			if (  solution[j + i * N] != C[j + i * N] ) {
				std::cout << i << ", " << j << ", " << solution[j + i * N] << ", " << C[j + i * N] << std::endl;
				std::cout << "Values are different !" << std::endl;
			}
		}

	return 0;
}
