#include <iostream>
#include <stdlib.h>
#include <cuAlgo.hpp>

int main() {

	hipError_t err;
	size_t K = 1024;
	size_t N = 1024;

	int * R = (int *)malloc(K * N * sizeof(int));
	for (size_t i = 0 ; i < K ; ++i)
		for (size_t j = 0 ; j < N ; ++j)
			R [j + i * N] = j * i;

	int * V = (int *)malloc(K * N * sizeof(int));
	for (size_t i = 0 ; i < K ; ++i)
		for (size_t j = 0 ; j < N ; ++j)
			V [j + i * N] = N * K - j * i;

	int * C = (int *)malloc(N*K * sizeof(int));
	int * solution = (int *)malloc(N*K * sizeof(int));

	int *d_R;
	err = hipMalloc(&d_R, K * N * sizeof(int));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	int *d_V;
	err = hipMalloc(&d_V, K * N * sizeof(int));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	int *d_C;
	err = hipMalloc(&d_C, N * K * sizeof(int));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy ( d_R, R, K * N *sizeof(int), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy ( d_V, V, K * N *sizeof(int), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	std::cout << "launching kernels ..." << std::endl;
	for (size_t i = 0; i < 5; ++i)
		convolution1dMatrix(d_R, d_V, d_C, N, K);
	std::cout << "launching kernels done ..." << std::endl;

	for (size_t i = 0 ; i < N ; ++i)
		solution[i] = 0;

	for (int j = 0 ; j < K ; ++j) {

		solution[j * N] = R[j * N] * V[j * N];

		for (int i = 1; i < N / 2; ++i)
			solution[i + j * N] = R[i + j * N] * V[i + j * N] -
			                      R[N - i + j * N] * V[N - i + j * N];

		solution[N / 2 + j * N] = R[N / 2 + j * N] * V[N / 2 + j * N];

		for (int i = N / 2 + 1, k = 0; i < N; ++i, ++k)
			solution[i + j * N] = R[N / 2 - 1 - k + j * N] * V[i + j * N] +
			                      R[i + j * N] * V[N / 2 - 1 - k + j * N];
	}

	err = hipMemcpy ( C, d_C, N * K * sizeof(int), hipMemcpyDeviceToHost );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMemcpy): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t j = 0 ; j < K ; ++j)
		for (size_t i = 0 ; i < N ; ++i)
			if (solution[i + j * N] != C[i + j * N]) {
				std::cout << "Values different" << std::endl;
				exit(EXIT_FAILURE);
			}

	return 0;
}
