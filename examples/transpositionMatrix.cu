/*
 * @file transpositionMatrix.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */
#include <iostream>
#include <stdlib.h>
#include <cuAlgo.hpp>

int main() {

	hipError_t err;
	unsigned int size_x = 1024;
	unsigned int size_y = 1024;
	float * input  = (float *)malloc(size_x * size_y * sizeof(float));
	float * output = (float *)malloc(size_x * size_y * sizeof(float));
	for(unsigned int j = 0; j < size_y; ++j)
		for (unsigned int i = 0; i < size_x ; ++i)
		input[i + j * size_x] = i + j * size_x;

	float *d_input;
	err = hipMalloc(&d_input, size_x * size_y * sizeof(float));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	float *d_output;
	err = hipMalloc(&d_output, size_x * size_y * sizeof(float));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy ( d_input, input, size_x * size_y * sizeof(float), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
	
	for (int i = 0; i < 5; ++i)
		transposeMatrix(d_input, d_output, size_x, size_y);

	err = hipMemcpy ( output, d_output, size_x * size_y * sizeof(float), hipMemcpyDeviceToHost );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	for(int j = 0; j < size_y; ++j)
		for (int i = 0; i < size_x ; ++i)
			if (input[i + j * size_x] != output[j + i * size_y]) {
				std::cout << "Values different" << std::endl;
				return 1;
			}

	return 0;
}