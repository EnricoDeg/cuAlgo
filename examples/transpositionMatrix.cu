#include <iostream>
#include <stdlib.h>
#include <cuAlgo.hpp>

int main() {

	hipError_t err;
	unsigned int size_x = 1024;
	unsigned int size_y = 1024;
	float * input  = (float *)malloc(size_x * size_y * sizeof(float));
	float * output = (float *)malloc(size_x * size_y * sizeof(float));
	for(unsigned int j = 0; j < size_y; ++j)
		for (unsigned int i = 0; i < size_x ; ++i)
		input[i + j * size_x] = i + j * size_x;

	float *d_input;
	err = hipMalloc(&d_input, size_x * size_y * sizeof(float));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	float *d_output;
	err = hipMalloc(&d_output, size_x * size_y * sizeof(float));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy ( d_input, input, size_x * size_y * sizeof(float), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
	
	for (int i = 0; i < 5; ++i)
		transposeMatrix(d_input, d_output, size_x, size_y);

	err = hipMemcpy ( output, d_output, size_x * size_y * sizeof(float), hipMemcpyDeviceToHost );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	for(int j = 0; j < size_y; ++j)
		for (int i = 0; i < size_x ; ++i)
			if (input[i + j * size_x] != output[j + i * size_y])
				return 1;

	return 0;
}