/*
 * @file reductionVector.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */
#include <iostream>
#include <stdlib.h>
#include <cuAlgo.hpp>

int main() {

	hipError_t err;
	int nblocks = 4096;
	int size = 1024*nblocks;
	int * input = (int *)malloc(size * sizeof(int));
	int * output = (int *)malloc(sizeof(int));
	for(int i = 0; i < nblocks; ++i)
		for (int j = 0; j < 1024 ; ++j)
		input[j + i*1024] = j;

	int *d_input;
	err = hipMalloc(&d_input, size*sizeof(int));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	int *d_output;
	err = hipMalloc(&d_output, sizeof(int));
	if (err != hipSuccess) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy ( d_input, input, (size_t)size*sizeof(int), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
	
	for (int i = 0; i < 5; ++i)
		reduce1dVector(d_input, d_output, size);

	output[0] = 0;
	for(int i = 0; i < size; ++i)
		output[0] += input[i];

	std::cout << "CPU solution = " << output[0] << std::endl;

	err = hipMemcpy ( output, d_output, sizeof(int), hipMemcpyDeviceToHost );
	if ( err != hipSuccess ) {
		std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

	std::cout << "GPU solution = " << output[0] << std::endl;

	return 0;
}
