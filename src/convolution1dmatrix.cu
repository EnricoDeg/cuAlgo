#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuAlgo.hpp"
#include <chrono>
#include "utils.h"

using namespace std::chrono;

#define THREADS_PER_BLOCK 1024
#define COMPUTE_PER_THREAD  32

__global__ void convolution1dMatrixKernel(const int *__restrict__ R,
                                          const int *__restrict__ V,
                                                int *__restrict__ C,
                                          size_t                  N,
                                          size_t                  K) {

	const size_t tid = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (tid > N * K)
		return;

	const size_t col  = tid % N;
	const size_t row  = tid / N;

	if (col == 0 || col == N/2) {

		C[col + N * row] = R[col + N * row] * V[col + N * row];
	} else if (col > 0 && col < N /2) {

		C[col + N * row] = R[col + N * row] * V[col + N * row] -
		                   R[N - col + N * row] * V[N - col + N * row] ;
	} else {

		size_t j = col - ( N / 2 + 1 );
		C[col + N * row] = R[N / 2 - 1 - j + N * row] * V[col + N * row] +
		                   R[col + N * row] * V[N / 2 - 1 - j + N * row] ;
	}

}

__global__ void convolution1dMatrixKernel1(const int *__restrict__ R,
                                           const int *__restrict__ V,
                                                 int *__restrict__ C,
                                           size_t                  N,
                                           size_t                  K,
                                           size_t             chunks) {

	const size_t tid = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

	if (tid > N / 2 * chunks)
		return;

#pragma unroll
	for (size_t i = 0; i < K / chunks; ++i) {

		size_t col = ( i * N / 2 * chunks + tid ) % ( N / 2 );
		const size_t row = ( i * N / 2 * chunks + tid ) / ( N / 2 );

		if (col == 0) {

			C[col + N * row] = R[col + N * row] * V[col + N * row];
			C[col + N / 2 + N * row] = R[col + N / 2 + N * row] * V[col + N / 2 + N * row];
		} else if (col > 0 && col < N /2) {

			C[col + N * row] = R[col + N * row] * V[col + N * row] -
			                   R[N - col + N * row] * V[N - col + N * row] ;
			col += N / 2;
			size_t j = col - ( N / 2 + 1 );
			C[col + N * row] = R[N / 2 - 1 - j + N * row] * V[col + N * row] +
			                   R[col + N * row] * V[N / 2 - 1 - j + N * row] ;
		}
	}
}

void convolution1dMatrix(int *  R,
                         int *  V,
                         int *  C,
                         size_t N,
                         size_t K) {

	size_t chunks = K / COMPUTE_PER_THREAD;
	std::cout << "chunks = " << chunks << std::endl;
	std::cout << "K = " << K << std::endl;

	dim3 block(THREADS_PER_BLOCK);
	dim3 grid(div_ceil(N / 2 * chunks, THREADS_PER_BLOCK));

	std::cout << "threadsPerBlock = " << THREADS_PER_BLOCK << std::endl;
	std::cout << "blocksPerGrid   = " << div_ceil(N * chunks, THREADS_PER_BLOCK) << std::endl;

	auto start = high_resolution_clock::now();
	convolution1dMatrixKernel1<<<grid, block>>>(R, V, C, N, K, chunks);
	check_cuda( hipDeviceSynchronize() );
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;
}
