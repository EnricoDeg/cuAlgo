#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuAlgo.hpp"
#include <chrono>

using namespace std::chrono;

__device__ void warpReduce(volatile int* sdata, unsigned int tid) {
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
}

__global__ void reduce1dKernel(int *g_idata, int *g_odata) {

	// use dynamic shared memory
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid < 32) warpReduce(sdata, tid);

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce1dKernelFlexible(int *g_idata, int *g_odata) {

	// use dynamic shared memory
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void reduce1d(int *g_idata, int *g_odata, int size) {

	int threadsPerBlock = size > 1024 ? 1024 : size/2;
	int blocksPerGrid = size / (2*threadsPerBlock) + (size % (2*threadsPerBlock) > 0);
	std::cout << "threadsPerBlock = " << threadsPerBlock << std::endl;
	std::cout << "blocksPerGrid   = " << blocksPerGrid   << std::endl;
	dim3 blocksPerGrid3(blocksPerGrid, 1, 1);
	dim3 threadsPerBlock3(threadsPerBlock, 1, 1);

	if (blocksPerGrid == 1) {

		auto start = high_resolution_clock::now();
		reduce1dKernelFlexible<<<blocksPerGrid3, threadsPerBlock3, (size_t)threadsPerBlock*sizeof(int)>>>(g_idata, g_odata);
		hipError_t err = hipDeviceSynchronize();
		if ( err != hipSuccess ) {
			std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;
	} else {

		int * d_buffer;
		hipError_t err = hipMalloc(&d_buffer, blocksPerGrid*sizeof(int));
		if (err != hipSuccess) {
			std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto start = high_resolution_clock::now();
		reduce1dKernel<<<blocksPerGrid3, threadsPerBlock3, (size_t)threadsPerBlock*sizeof(int)>>>(g_idata, d_buffer);
		err = hipDeviceSynchronize();
		if ( err != hipSuccess ) {
			std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;

		reduce1d(d_buffer, g_odata, blocksPerGrid);
		err = hipFree ( d_buffer );
		if ( err != hipSuccess ) {
			fprintf(stderr, "CUDA error (hipFree): %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	}
}
