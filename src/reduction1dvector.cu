#include "hip/hip_runtime.h"
/*
 * @file reduction1dvector.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */
#include <iostream>
#include "cuAlgo.hpp"
#include "utils.hpp"
#include <chrono>

using namespace std::chrono;

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid +  8];
	if (blockSize >= 8)  sdata[tid] += sdata[tid +  4];
	if (blockSize >= 4)  sdata[tid] += sdata[tid +  2];
	if (blockSize >= 2)  sdata[tid] += sdata[tid +  1];
}

template <unsigned int blockSize>
__global__ void reduce1dKernel(int *g_idata, int *g_odata, unsigned int n) {

	// use dynamic shared memory
	extern __shared__ int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	// load to shared memory
	sdata[tid] = 0;
	while (i < n) {
		sdata[tid] += g_idata[i] + g_idata[i+blockSize];
		i += gridSize;
	}
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 1024) {
		if (tid < 512) {
			sdata[tid] += sdata[tid + 512];
		}
		__syncthreads();
	}
	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	
	if (tid < 32) warpReduce<blockSize>(sdata, tid);

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce1dKernelFlexible(int *g_idata, int *g_odata) {

	// use dynamic shared memory
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void reduce1dVector(int          *g_idata,
                    int          *g_odata,
                    int           size   ,
                    hipStream_t  stream ,
                    bool          async  ) {

	int threadsPerBlock = size > 1024 ? 1024 : size/2;
	int blocksPerGrid = size / (2*threadsPerBlock) + (size % (2*threadsPerBlock) > 0);
	unsigned int shmem = threadsPerBlock*sizeof(int);

	if (blocksPerGrid == 1) {

		dim3 blocksPerGrid3(blocksPerGrid, 1, 1);
		dim3 threadsPerBlock3(threadsPerBlock, 1, 1);
		print_kernel_config(threadsPerBlock, blocksPerGrid);

		TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
		     reduce1dKernelFlexible,
		     g_idata, g_odata);

	} else {

		dim3 blocksPerGrid3(blocksPerGrid, 1, 1);
		dim3 threadsPerBlock3(threadsPerBlock, 1, 1);
		print_kernel_config(threadsPerBlock, blocksPerGrid);

		int * d_buffer;
		check_cuda( hipMalloc(&d_buffer, blocksPerGrid*sizeof(int)) );
		switch (threadsPerBlock) {
			case 1024:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<1024>,
			     g_idata, d_buffer, size);
			break;
			case 512:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel< 512>,
			     g_idata, d_buffer, size);
			break;
			case 256:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel< 256>,
			     g_idata, d_buffer, size);
			break;
			case 128:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel< 128>,
			     g_idata, d_buffer, size);
			break;
			case 64:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<  64>,
			     g_idata, d_buffer, size);
			break;
			case 32:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel< 128>,
			     g_idata, d_buffer, size);
			break;
			case 16:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<  16>,
			     g_idata, d_buffer, size);
			break;
			case 8:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<   8>,
			     g_idata, d_buffer, size);
			break;
			case 4:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<   4>,
			     g_idata, d_buffer, size);
			break;
			case 2:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<   2>,
			     g_idata, d_buffer, size);
			break;
			case 1:
			TIME(blocksPerGrid3, threadsPerBlock3, shmem, stream, async,
			     reduce1dKernel<   1>,
			     g_idata, d_buffer, size);
			break;
		}

		reduce1dVector(d_buffer, g_odata, blocksPerGrid, stream, async);
		check_cuda( hipFree ( d_buffer ) );
	}
}
