#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuAlgo.hpp"
#include <chrono>

using namespace std::chrono;

__global__ void reduce1dKernel(int *g_idata, int *g_odata) {

	// use dynamic shared memory
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void reduce1d(int *g_idata, int *g_odata, int size) {

	int threadsPerBlock = size > 1024 ? 512 : size;
	int blocksPerGrid = size / threadsPerBlock + (size % threadsPerBlock > 0);
	std::cout << "threadsPerBlock = " << threadsPerBlock << std::endl;
	std::cout << "blocksPerGrid   = " << blocksPerGrid   << std::endl;
	dim3 blocksPerGrid3(blocksPerGrid, 1, 1);
	dim3 threadsPerBlock3(threadsPerBlock, 1, 1);

	if (blocksPerGrid == 1) {

		auto start = high_resolution_clock::now();
		reduce1dKernel<<<blocksPerGrid3, threadsPerBlock3, (size_t)threadsPerBlock*sizeof(int)>>>(g_idata, g_odata);
		hipError_t err = hipDeviceSynchronize();
		if ( err != hipSuccess ) {
			std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;
	} else {

		int * d_buffer;
		hipError_t err = hipMalloc(&d_buffer, blocksPerGrid*sizeof(int));
		if (err != hipSuccess) {
			std::cout << "CUDA error (hipMalloc): " <<  hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto start = high_resolution_clock::now();
		reduce1dKernel<<<blocksPerGrid3, threadsPerBlock3, (size_t)threadsPerBlock*sizeof(int)>>>(g_idata, d_buffer);
		err = hipDeviceSynchronize();
		if ( err != hipSuccess ) {
			std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;

		reduce1d(d_buffer, g_odata, blocksPerGrid);
		err = hipFree ( d_buffer );
		if ( err != hipSuccess ) {
			fprintf(stderr, "CUDA error (hipFree): %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	}
}
