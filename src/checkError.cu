#include <iostream>
#include <hip/hip_runtime.h>
#include "cuAlgo.hpp"
#include <chrono>

void check_cuda(hipError_t error) {

	if ( error != hipSuccess ) {
		std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
		exit(EXIT_FAILURE);
	}
}